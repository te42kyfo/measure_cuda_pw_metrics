#include "hip/hip_runtime.h"
#include <cupti_target.h>
#include <cupti_profiler_target.h>
#include <nvperf_host.h>
#include <hip/hip_runtime.h>
#include <string>
#include <stdio.h>
#include <stdlib.h>
#include <functional>
#include "Metric.hpp"
#include "Eval.hpp"
//#include <FileOp.h>

#define NVPW_API_CALL(apiFuncCall)                                             \
do {                                                                           \
    NVPA_Status _status = apiFuncCall;                                         \
    if (_status != NVPA_STATUS_SUCCESS) {                                      \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define CUPTI_API_CALL(apiFuncCall)                                            \
do {                                                                           \
    CUptiResult _status = apiFuncCall;                                         \
    if (_status != CUPTI_SUCCESS) {                                            \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define DRIVER_API_CALL(apiFuncCall)                                           \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                            \
    if (_status != hipSuccess) {                                             \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(-1);                                                              \
    }                                                                          \
} while (0)

static int numRanges = 4;
#define METRIC_NAME "smsp__warps_launched.avg+"

// Device code
  __global__ void VecAdd(const int* A, const int* B, int* C, int N)
 {
     int i = blockDim.x * blockIdx.x + threadIdx.x;
     if (i < N)
         C[i] = A[i] + B[i];
 }


  __global__ void VecMul(const int* A, const int* B, int* C, int N)
 {
     int i = blockDim.x * blockIdx.x + threadIdx.x;
     if (i < N)
         C[i] = A[i] * B[i];
 }

 // Device code
  __global__ void VecSub(const int* A, const int* B, int* C, int N)
 {
     int i = blockDim.x * blockIdx.x + threadIdx.x;
     if (i < N)
         C[i] = A[i] - B[i];
 }



static double VectorAddSubtract()
{
  int N = 500000000;
  size_t size = N * sizeof(int);
  int threadsPerBlock = 0;
  int blocksPerGrid = 0;
  int *d_A, *d_B, *d_C, *d_D, *d_E;

  // Allocate vectors in device memory
  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);
  hipMalloc((void**)&d_D, size);
  hipMalloc((void**)&d_E, size);


  // Invoke kernel
  threadsPerBlock = 256;
  blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  printf("Launching kernel: blocks %d, thread/block %d\n",
         blocksPerGrid, threadsPerBlock);


  VecMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_E, N);
  VecSub<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_D, N);
  VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_D);
  hipFree(d_E);
  return 0.0;
}

bool CreateCounterDataImage(
    std::vector<uint8_t>& counterDataImage,
    std::vector<uint8_t>& counterDataScratchBuffer,
    std::vector<uint8_t>& counterDataImagePrefix)
{

    CUpti_Profiler_CounterDataImageOptions counterDataImageOptions;
    counterDataImageOptions.pCounterDataPrefix = &counterDataImagePrefix[0];
    counterDataImageOptions.counterDataPrefixSize = counterDataImagePrefix.size();
    counterDataImageOptions.maxNumRanges = numRanges;
    counterDataImageOptions.maxNumRangeTreeNodes = numRanges;
    counterDataImageOptions.maxRangeNameLength = 64;

    CUpti_Profiler_CounterDataImage_CalculateSize_Params calculateSizeParams = {CUpti_Profiler_CounterDataImage_CalculateSize_Params_STRUCT_SIZE};

    calculateSizeParams.pOptions = &counterDataImageOptions;
    calculateSizeParams.sizeofCounterDataImageOptions = CUpti_Profiler_CounterDataImageOptions_STRUCT_SIZE;

    CUPTI_API_CALL(cuptiProfilerCounterDataImageCalculateSize(&calculateSizeParams));

    CUpti_Profiler_CounterDataImage_Initialize_Params initializeParams = {CUpti_Profiler_CounterDataImage_Initialize_Params_STRUCT_SIZE};
    initializeParams.sizeofCounterDataImageOptions = CUpti_Profiler_CounterDataImageOptions_STRUCT_SIZE;
    initializeParams.pOptions = &counterDataImageOptions;
    initializeParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;

    counterDataImage.resize(calculateSizeParams.counterDataImageSize);
    initializeParams.pCounterDataImage = &counterDataImage[0];
    CUPTI_API_CALL(cuptiProfilerCounterDataImageInitialize(&initializeParams));

    CUpti_Profiler_CounterDataImage_CalculateScratchBufferSize_Params scratchBufferSizeParams = {CUpti_Profiler_CounterDataImage_CalculateScratchBufferSize_Params_STRUCT_SIZE};
    scratchBufferSizeParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;
    scratchBufferSizeParams.pCounterDataImage = initializeParams.pCounterDataImage;
    CUPTI_API_CALL(cuptiProfilerCounterDataImageCalculateScratchBufferSize(&scratchBufferSizeParams));

    counterDataScratchBuffer.resize(scratchBufferSizeParams.counterDataScratchBufferSize);

    CUpti_Profiler_CounterDataImage_InitializeScratchBuffer_Params initScratchBufferParams = {CUpti_Profiler_CounterDataImage_InitializeScratchBuffer_Params_STRUCT_SIZE};
    initScratchBufferParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;

    initScratchBufferParams.pCounterDataImage = initializeParams.pCounterDataImage;
    initScratchBufferParams.counterDataScratchBufferSize = scratchBufferSizeParams.counterDataScratchBufferSize;
    initScratchBufferParams.pCounterDataScratchBuffer = &counterDataScratchBuffer[0];

    CUPTI_API_CALL(cuptiProfilerCounterDataImageInitializeScratchBuffer(&initScratchBufferParams));

    return true;
}

bool runTest(std::function<double()> runPass, hipDevice_t cuDevice,
             std::vector<uint8_t>& configImage,
             std::vector<uint8_t>& counterDataScratchBuffer,
             std::vector<uint8_t>& counterDataImage,
             CUpti_ProfilerReplayMode profilerReplayMode,
             CUpti_ProfilerRange profilerRange)
{

    hipCtx_t cuContext;
    DRIVER_API_CALL(hipCtxCreate(&cuContext, 0, cuDevice));

    CUpti_Profiler_BeginSession_Params beginSessionParams = {CUpti_Profiler_BeginSession_Params_STRUCT_SIZE};
    CUpti_Profiler_SetConfig_Params setConfigParams = {CUpti_Profiler_SetConfig_Params_STRUCT_SIZE};
    CUpti_Profiler_EnableProfiling_Params enableProfilingParams = {CUpti_Profiler_EnableProfiling_Params_STRUCT_SIZE};
    CUpti_Profiler_DisableProfiling_Params disableProfilingParams = {CUpti_Profiler_DisableProfiling_Params_STRUCT_SIZE};

    beginSessionParams.ctx = NULL;
    beginSessionParams.counterDataImageSize = counterDataImage.size();
    beginSessionParams.pCounterDataImage = &counterDataImage[0];
    beginSessionParams.counterDataScratchBufferSize = counterDataScratchBuffer.size();
    beginSessionParams.pCounterDataScratchBuffer = &counterDataScratchBuffer[0];
    beginSessionParams.range = profilerRange;
    beginSessionParams.replayMode = profilerReplayMode;
    beginSessionParams.maxRangesPerPass = numRanges;
    beginSessionParams.maxLaunchesPerPass = numRanges;

    CUPTI_API_CALL(cuptiProfilerBeginSession(&beginSessionParams));

    setConfigParams.pConfig = &configImage[0];
    setConfigParams.configSize = configImage.size();

    if(profilerReplayMode == CUPTI_KernelReplay)    /* Profile in KernelReplayMode */
    {
        setConfigParams.passIndex = 0;
        CUPTI_API_CALL(cuptiProfilerSetConfig(&setConfigParams));
        CUPTI_API_CALL(cuptiProfilerEnableProfiling(&enableProfilingParams));
        runPass();
        CUPTI_API_CALL(cuptiProfilerDisableProfiling(&disableProfilingParams));
    }
    else if(profilerReplayMode == CUPTI_UserReplay) /* Profiler in UserReplayMode */
    {
        setConfigParams.passIndex = 0;
        CUPTI_API_CALL(cuptiProfilerSetConfig(&setConfigParams));
        /* User takes the resposiblity of replaying the kernel launches */
        CUpti_Profiler_BeginPass_Params beginPassParams = {CUpti_Profiler_BeginPass_Params_STRUCT_SIZE};
        CUpti_Profiler_EndPass_Params endPassParams = {CUpti_Profiler_EndPass_Params_STRUCT_SIZE};
        do
        {
            CUPTI_API_CALL(cuptiProfilerBeginPass(&beginPassParams));
            {
                CUPTI_API_CALL(cuptiProfilerEnableProfiling(&enableProfilingParams));
                runPass();
                CUPTI_API_CALL(cuptiProfilerDisableProfiling(&disableProfilingParams));
            }
            CUPTI_API_CALL(cuptiProfilerEndPass(&endPassParams));
        }while(!endPassParams.allPassesSubmitted);
        CUpti_Profiler_FlushCounterData_Params flushCounterDataParams = {CUpti_Profiler_FlushCounterData_Params_STRUCT_SIZE};
        CUPTI_API_CALL(cuptiProfilerFlushCounterData(&flushCounterDataParams));
    }
    CUpti_Profiler_UnsetConfig_Params unsetConfigParams = {CUpti_Profiler_UnsetConfig_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerUnsetConfig(&unsetConfigParams));
    CUpti_Profiler_EndSession_Params endSessionParams = {CUpti_Profiler_EndSession_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerEndSession(&endSessionParams));

    DRIVER_API_CALL(hipCtxDestroy(cuContext));

    return true;
}

double measureMetric(std::function<double()> runPass, std::vector<std::string> metricNames) {
    hipFree(0);

    int deviceNum = 0;
    hipDevice_t cuDevice;
    int computeCapabilityMajor = 0, computeCapabilityMinor = 0;
    DRIVER_API_CALL(hipDeviceGet(&cuDevice, deviceNum));
    DRIVER_API_CALL(hipDeviceGetAttribute(&computeCapabilityMajor, hipDeviceAttributeComputeCapabilityMajor, cuDevice));
    DRIVER_API_CALL(hipDeviceGetAttribute(&computeCapabilityMinor, hipDeviceAttributeComputeCapabilityMinor, cuDevice));
    if(computeCapabilityMajor < 7) {
      printf("Sample unsupported on Device with compute capability < 7.0\n");
      return -2.0;
    }

    std::vector<uint8_t> counterDataImagePrefix;
    std::vector<uint8_t> configImage;
    std::vector<uint8_t> counterDataImage;
    std::vector<uint8_t> counterDataScratchBuffer;

    CUpti_Profiler_Initialize_Params profilerInitializeParams = {CUpti_Profiler_Initialize_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerInitialize(&profilerInitializeParams));
    /* Get chip name for the cuda  device */
    CUpti_Device_GetChipName_Params getChipNameParams = { CUpti_Device_GetChipName_Params_STRUCT_SIZE };
    getChipNameParams.deviceIndex = deviceNum;
    CUPTI_API_CALL(cuptiDeviceGetChipName(&getChipNameParams));
    std::string chipName(getChipNameParams.pChipName);

    /* Generate configuration for metrics, this can also be done offline*/
    NVPW_InitializeHost_Params initializeHostParams = { NVPW_InitializeHost_Params_STRUCT_SIZE };
    NVPW_API_CALL(NVPW_InitializeHost(&initializeHostParams));
    if (metricNames.size()) {
        if(!NV::Metric::Config::GetConfigImage(chipName, metricNames, configImage)) {
            std::cout << "Failed to create configImage" << std::endl;
            return -1.0;
        }
        if(!NV::Metric::Config::GetCounterDataPrefixImage(chipName, metricNames, counterDataImagePrefix)) {
            std::cout << "Failed to create counterDataImagePrefix" << std::endl;
            return -1.0;
        }
    } else {
        std::cout << "No metrics provided to profile" << std::endl;
        return -1.0;
    }




    CUpti_ProfilerReplayMode profilerReplayMode = CUPTI_KernelReplay;
    CUpti_ProfilerRange profilerRange = CUPTI_AutoRange;

    if(!CreateCounterDataImage(counterDataImage, counterDataScratchBuffer, counterDataImagePrefix)) {
        std::cout << "Failed to create counterDataImage" << std::endl;
    }
    runTest(runPass,  cuDevice, configImage, counterDataScratchBuffer, counterDataImage, profilerReplayMode, profilerRange);


    CUpti_Profiler_DeInitialize_Params profilerDeInitializeParams = {CUpti_Profiler_DeInitialize_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerDeInitialize(&profilerDeInitializeParams));

    NV::Metric::Eval::PrintMetricValues(chipName, counterDataImage, metricNames);

    return 0.0;
}

int main(int argc, char* argv[]) {
    measureMetric(VectorAddSubtract, { "dram__bytes_write.sum.per_second", "dram__bytes_read.sum.per_second"});



    return 0;
}
